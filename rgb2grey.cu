#include <isostream>
#include <string>
#include <cassert>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv/opencv.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

cv::Mat imageRGBA;
cv::Mat imageGrey;
